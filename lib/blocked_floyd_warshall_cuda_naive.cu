#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern "C" {
    #include "blocked_floyd_warshall.h"
}

__forceinline__
__host__ int check_cuda_error(t_error* err) {
	hipError_t errCode = hipPeekAtLastError();
	if (errCode != hipSuccess) {
		err->err_code = errCode;
		snprintf(err->err_msg, sizeof(err->err_msg), "%s", hipGetErrorString(errCode));
		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}

__forceinline__
__device__ void calc(int* graph, int n, int k, int i, int j) {
	if ((i >= n) || (j >= n) || (k >= n)) return;
	const unsigned int kj = k * n + j;
	const unsigned int ij = i * n + j;
	const unsigned int ik = i * n + k;
	int t1 = graph[ik] + graph[kj];
	int t2 = graph[ij];
	graph[ij] = (t1 < t2) ? t1 : t2;
}

__global__ void floyd_warshall_naive_kernel(int n, int k, int* graph) {
	const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
	calc(graph, n, k, i, j);
}

extern "C"
__host__ int blocked_floyd_warshall(int* graph, int n, int b, int* apsp, t_error* err) {

	if (n <= 0) {
		err->err_code = WRONG_NUM_OF_NODES_ERR;
		char err_msg[100] = {
				"The number of nodes should be > 0"};
		snprintf(err->err_msg, sizeof(err->err_msg), "%s", err_msg);
		return EXIT_FAILURE;
	}

	if (b <= 0 || b > n) {
		err->err_code = WRONG_BLOCK_SIZE_ERR;
		char err_msg[100] = {
				"The number of blocks b should be: 0 < b <= n, where n is the number of nodes"};
		snprintf(err->err_msg, sizeof(err->err_msg), "%s", err_msg);
		return EXIT_FAILURE;
	}
	const size_t size = n * n * sizeof(int);
	int* device_graph;

	hipMalloc(&device_graph, size);
	hipMemcpy(device_graph, graph,  size, hipMemcpyHostToDevice);

	dim3 block_dim(b, b, 1);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x,
                  (n + block_dim.y - 1) / block_dim.y);

    int k;
	for (k = 0; k < n; k++) {
		floyd_warshall_naive_kernel<<<grid_dim, block_dim>>>(n, k, device_graph);
		hipDeviceSynchronize();
	}

	hipMemcpy(apsp, device_graph, size, hipMemcpyDeviceToHost);
	hipFree(device_graph);

	return check_cuda_error(err);
}
